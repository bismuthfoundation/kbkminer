#include "hip/hip_runtime.h"
#include "bis.h"

#define SHA2_SHFR(x, n)    (x >> n)
#define SHA2_ROTR(x, n)   ((x >> n) | (x << ((sizeof(x) << 3) - n)))
#define SHA2_ROTL(x, n)   ((x << n) | (x >> ((sizeof(x) << 3) - n)))
#define SHA2_CH(x, y, z)        ((x & (y ^ z)) ^ z) //form https://github.com/leocalm/Lyra/blob/master/GPU_attacks/yescryptCUDA/sha256.cu
#define SHA2_MAJ(x, y, z) ((x & (y | z)) | (y & z)) //form https://github.com/leocalm/Lyra/blob/master/GPU_attacks/yescryptCUDA/sha256.cu

char const hex_chars_small_h[] = { '0','1','2','3','4','5','6','7','8','9','a','b','c','d','e','f' };

__device__ __host__ __forceinline__ uint32_t rotr(uint32_t x, uint32_t n) {
#ifdef  __CUDA_ARCH__
        uint32_t result;
        asm("shf.r.wrap.b32  %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(x), "r"(n));
        return result;
#else
        return SHA2_ROTR(x, n);
#endif
}

#define SHA256_F1(x) (rotr(x,  2) ^ rotr(x, 13) ^ rotr(x, 22))
#define SHA256_F2(x) (rotr(x,  6) ^ rotr(x, 11) ^ rotr(x, 25))
#define SHA256_F3(x) (rotr(x,  7) ^ rotr(x, 18) ^ SHA2_SHFR(x,  3))
#define SHA256_F4(x) (rotr(x, 17) ^ rotr(x, 19) ^ SHA2_SHFR(x, 10))

#define SHA2_PACK32(str, x)                     \
{                                               \
    *(x) =   ((uint32_t) *((str) + 3)      )    \
           | ((uint32_t) *((str) + 2) <<  8)    \
           | ((uint32_t) *((str) + 1) << 16)    \
           | ((uint32_t) *((str) + 0) << 24);   \
}

__device__ __constant__ uint32_t const sha256_k[64] = {
        0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
        0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
        0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
        0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
        0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
        0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
        0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
        0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
        0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
        0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
        0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
        0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
        0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
        0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

uint32_t const sha256_k_cpu[64] = {
        0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
        0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
        0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
        0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
        0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
        0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
        0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
        0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
        0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
        0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
        0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
        0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
        0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
        0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

template<class T, class T2>
__device__ __host__ __forceinline__ void memset_simple12(T* arr, T2 val) {
#pragma unroll 12
        for (uint32_t i = 0; i < 12; i++) {
                arr[i] = val;
        }
}

__device__ __host__ __forceinline__ void sha224_init(uint32_t* m_h) {
        m_h[0] = 0xc1059ed8;
        m_h[1] = 0x367cd507;
        m_h[2] = 0x3070dd17;
        m_h[3] = 0xf70e5939;
        m_h[4] = 0xffc00b31;
        m_h[5] = 0x68581511;
        m_h[6] = 0x64f98fa7;
        m_h[7] = 0xbefa4fa4;
}

__device__  __forceinline__ void transform(const uint32_t* message_int, uint32_t* m_h) {
        uint32_t w[64];
        uint32_t wv[8];
        uint32_t t1, t2;

#pragma unroll 16
        for (uint32_t j = 0; j < 16; j++) {
                w[j] = message_int[j];
        }

#pragma unroll 48
        for (uint32_t j = 16; j < 64; j++) {
                w[j] = SHA256_F4(w[j - 2]) + w[j - 7] + SHA256_F3(w[j - 15]) + w[j - 16];
        }
#pragma unroll 8
        for (uint32_t j = 0; j < 8; j++) {
                wv[j] = m_h[j];
        }
#pragma unroll 64
        for (uint32_t j = 0; j < 64; j++) {
                t1 = wv[7] + SHA256_F2(wv[4]) + SHA2_CH(wv[4], wv[5], wv[6])
                        + sha256_k[j] + w[j];
                t2 = SHA256_F1(wv[0]) + SHA2_MAJ(wv[0], wv[1], wv[2]);
                wv[7] = wv[6];
                wv[6] = wv[5];
                wv[5] = wv[4];
                wv[4] = wv[3] + t1;
                wv[3] = wv[2];
                wv[2] = wv[1];
                wv[1] = wv[0];
                wv[0] = t1 + t2;
        }
#pragma unroll 8
        for (uint32_t j = 0; j < 8; j++) {
                m_h[j] += wv[j];
        }
}

void transform_cpu(const uint32_t* message_int, uint32_t* m_h) {
        uint32_t w[64];
        uint32_t wv[8];
        uint32_t t1, t2;

        for (uint32_t j = 0; j < 16; j++) {
                w[j] = message_int[j];
        }

        for (uint32_t j = 16; j < 64; j++) {
                w[j] = SHA256_F4(w[j - 2]) + w[j - 7] + SHA256_F3(w[j - 15]) + w[j - 16];
        }

        for (uint32_t j = 0; j < 8; j++) {
                wv[j] = m_h[j];
        }

        for (uint32_t j = 0; j < 64; j++) {
                t1 = wv[7] + SHA256_F2(wv[4]) + SHA2_CH(wv[4], wv[5], wv[6])
                        + sha256_k_cpu[j] + w[j];
                t2 = SHA256_F1(wv[0]) + SHA2_MAJ(wv[0], wv[1], wv[2]);
                wv[7] = wv[6];
                wv[6] = wv[5];
                wv[5] = wv[4];
                wv[4] = wv[3] + t1;
                wv[3] = wv[2];
                wv[2] = wv[1];
                wv[1] = wv[0];
                wv[0] = t1 + t2;
        }

        for (uint32_t j = 0; j < 8; j++) {
                m_h[j] += wv[j];
        }
}

__device__  __forceinline__ void transform_sep_nonce(const uint32_t* message_int, const uint32_t* nonce, uint32_t* m_h) {
        uint32_t w[64];
        uint32_t wv[8];
        uint32_t t1, t2;

#pragma unroll 4
        for (int j = 0; j < 4; j++) {
                w[j] = message_int[j];
        }
#pragma unroll 10
        for (int j = 6; j < 16; j++) {
                w[j] = message_int[j];
        }

        w[4] = nonce[0];
        w[5] = nonce[1];

#pragma unroll 48
        for (int j = 16; j < 64; j++) {
                w[j] = SHA256_F4(w[j - 2]) + w[j - 7] + SHA256_F3(w[j - 15]) + w[j - 16];
        }
#pragma unroll 8
        for (int j = 0; j < 8; j++) {
                wv[j] = m_h[j];
        }
#pragma unroll 64
        for (int j = 0; j < 64; j++) {
                t1 = wv[7] + SHA256_F2(wv[4]) + SHA2_CH(wv[4], wv[5], wv[6])
                        + sha256_k[j] + w[j];
                t2 = SHA256_F1(wv[0]) + SHA2_MAJ(wv[0], wv[1], wv[2]);
                wv[7] = wv[6];
                wv[6] = wv[5];
                wv[5] = wv[4];
                wv[4] = wv[3] + t1;
                wv[3] = wv[2];
                wv[2] = wv[1];
                wv[1] = wv[0];
                wv[0] = t1 + t2;
        }
#pragma unroll 8
        for (int j = 0; j < 8; j++) {
                m_h[j] += wv[j];
        }
}

__device__  __forceinline__ void sha224_final(uint32_t* m_block, uint32_t* m_h) {
        memset_simple12(m_block + 16 / 4, 0);
        m_block[16 / 4] = m_block[16 / 4] | 0x80000000;
        m_block[60 / 4] = 1152;
        transform(m_block, m_h);
}

__device__  __forceinline__ void modNonce(const uint32_t* nonce, uint32_t* new_nonce, uint32_t const th_id) {
        char const hex_chars_small[] = { '0','1','2','3','4','5','6','7','8','9','a','b','c','d','e','f' };
        new_nonce[0] = 0;

        uint8_t chIdx = (uint8_t)(th_id & 0xF);
        uint32_t val = (uint32_t)hex_chars_small[chIdx];
        new_nonce[0] = new_nonce[0] | (val << 24);        // 0xXX000000

        chIdx = (uint8_t)((th_id >> 4) & 0xF);
        val = (uint32_t)hex_chars_small[chIdx];
        new_nonce[0] = new_nonce[0] | (val << 16); // 0xXXXX0000

        chIdx = (uint8_t)((th_id >> 8) & 0xF);
        val = (uint32_t)hex_chars_small[chIdx];
        new_nonce[0] = new_nonce[0] | (val << 8);  // 0xXXXXXX00

        chIdx = (uint8_t)((th_id >> 12) & 0xF);
        val = (uint32_t)hex_chars_small[chIdx];
        new_nonce[0] = new_nonce[0] | val;                // 0xXXXXXXXX


        new_nonce[1] = nonce[5] & 0x0000FFFF;
        chIdx = (uint8_t)((th_id >> 16) & 0xF);
        val = (uint32_t)hex_chars_small[chIdx];
        new_nonce[1] = new_nonce[1] | (val << 24);        // 0xXX00????

        chIdx = (uint8_t)((th_id >> 20) & 0xF);
        val = (uint32_t)hex_chars_small[chIdx];
        new_nonce[1] = new_nonce[1] | (val << 16);        // 0xXXXX????
}

void modNonceCpu(unsigned char* nonce, uint32_t const th_id) {
        unsigned char chIdx = (th_id & 0xF);
        uint8_t i = 24;

        chIdx = (uint8_t)(th_id & 0xF);
        nonce[i] = hex_chars_small_h[chIdx];      // 0xXX000000

        chIdx = (uint8_t)((th_id >> 4) & 0xF);
        i++;
        nonce[i] = hex_chars_small_h[chIdx]; // 0xXXXX0000

        chIdx = (uint8_t)((th_id >> 8) & 0xF);
        i++;
        nonce[i] = hex_chars_small_h[chIdx];  // 0xXXXXXX00

        chIdx = (uint8_t)((th_id >> 12) & 0xF);
        i++;
        nonce[i] = hex_chars_small_h[chIdx];              // 0xXXXXXXXX

        chIdx = (uint8_t)((th_id >> 16) & 0xF);
        i++;
        nonce[i] = hex_chars_small_h[chIdx];      // 0xXX00????

        chIdx = (uint8_t)((th_id >> 20) & 0xF);
        i++;
        nonce[i] = hex_chars_small_h[chIdx];      // 0xXXXX????
}

__device__ __forceinline__ void shiftLeft4Bytes7Ints(uint32_t* uintArr) {
#pragma unroll 6
        for (int i = 0; i < 6; i++) {
                uintArr[i] = uintArr[i] << 4;

                uintArr[i] = uintArr[i] | (uintArr[i + 1] >> 28);
        }
        uintArr[6] = uintArr[6] << 4;
}

__device__ __forceinline__ void anneal3(const uint32_t* map, uint32_t* hash) {
        int index = ((hash[6] & ~0x7) % RND_LEN) + 6;
#pragma unroll
        for (int i = 0; i < 7; ++i) {
                hash[i] ^= map[index - i];
        }
}

__global__ void sha224_find(const uint32_t* message_int, uint32_t* hash, uint32_t* mining_cond, const int32_t tail_length_in_chars, uint32_t* map, uint32_t* nonce_found_on_th_id) {
        uint32_t const one_pad_x_chars_lsb_constants_loc[] = { 0x0, 0xF, 0xFF, 0xFFF, 0xFFFF, 0xFFFFF, 0xFFFFFF, 0xFFFFFFF, 0xFFFFFFFF };
        uint32_t const th_id = blockIdx.x * blockDim.x + threadIdx.x;

        uint32_t nonce_local[2];
        __shared__ uint32_t message_int_16_to_36[20];
        uint32_t m_block[SHA224_256_BLOCK_SIZE / 4];
        uint32_t hash_local[8];
        uint32_t sum = 0;
        uint32_t rightPadOfHashForLastCondition = one_pad_x_chars_lsb_constants_loc[tail_length_in_chars];
        uint32_t mining_cond_local[N_32BITS_IN_MINING_CONDITION];


        if ((threadIdx.x > 15) && (threadIdx.x < 36)) {
                message_int_16_to_36[threadIdx.x - 16] = message_int[threadIdx.x];
        }
        __syncthreads();

        //Local copy of message
#pragma unroll 8
        for (int i = 0; i < 8; i++) { hash_local[i] = hash[i]; }

        //Randomize message by editing nonce
        modNonce((const uint32_t*)&message_int_16_to_36[0], nonce_local, th_id);

        //Update
        transform_sep_nonce((const uint32_t*)message_int_16_to_36, nonce_local, hash_local);

#pragma unroll 4
        for (int i = 0; i < 4; i++) {
                m_block[i] = message_int_16_to_36[16 + i];
        }
        //EO UPDATE

        //Final
        sha224_final(m_block, hash_local);


        //Anneal3
        anneal3(map, hash_local);
        //Eo anneal3

        //copy mining condition local
#pragma unroll 2
        for (uint32_t i = 0; i < 2; i++) { mining_cond_local[i] = mining_cond[i]; }

#pragma unroll 8
        for (int shifts = 0; shifts < 8; shifts++) {
#pragma unroll 6
                for (int h_i = 0; h_i < 6; h_i++) {
                        sum = 0;
                        sum = sum | (hash_local[h_i] ^ mining_cond_local[0]);
                        int h1 = (hash_local[h_i + 1] | rightPadOfHashForLastCondition);
                        sum = sum | (h1 ^ mining_cond_local[1]);
                        if (sum == 0) {
                                *nonce_found_on_th_id = th_id;
                        }
                }
                shiftLeft4Bytes7Ints(hash_local);
        }
}

std::string nonceFromThreadId(std::string nonce, uint32_t th_id) {
        unsigned char nonce_buf[32];
        for (int ii = 0; ii < 32; ii++) { nonce_buf[ii] = nonce[ii]; }
        modNonceCpu(nonce_buf, th_id);
        for (int ii = 0; ii < 32; ii++) { nonce[ii] = nonce_buf[ii]; }
        return nonce;
}

uint32_t StringToInts(std::string hex_str, uint32_t* ret, bool onePad = true) {
        //pad with 0's
        while (hex_str.length() % 8 != 0) {
                if (onePad) {
                        hex_str = hex_str + "f";
                }
                else {
                        hex_str = hex_str + "0";
                }
        }

        int len = 0;
        for (unsigned i = 0; i < hex_str.length(); i += 8) {
                std::string hex32_numb = hex_str.substr(i, 8);
                unsigned int x = std::stoul(hex32_numb, nullptr, 16);
                ret[len] = x;
                len++;
        }
        return len;
}

void cur(hipError_t error) {
        if (error != hipSuccess) {
                std::cout << hipGetErrorString(error);
                throw std::runtime_error(hipGetErrorString(error));
        }
}

int32_t BisCuda::GetNumDevices() {
        int32_t deviceCount = -1;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err == hipSuccess)
                return deviceCount;

        if (err == hipErrorInsufficientDriver) {
                int driverVersion = -1;
                hipDriverGetVersion(&driverVersion);
                if (driverVersion == 0)
                        throw std::runtime_error{ "No CUDA driver found" };
                throw std::runtime_error{ "Insufficient CUDA driver: " + std::to_string(driverVersion) };
        }

        throw std::runtime_error{ hipGetErrorString(err) };
}

void packMsg(const unsigned char* message, uint32_t* msg_int) {
        for (int j = 0; j < 144 / 4; j++) {
                SHA2_PACK32(&message[j << 2], &msg_int[j]);
        }
}

std::string BisCuda::GenerateNonce_DoNotCall() {
        std::string nonce = "";
        for (int i = 0; i < 32; i++) {

                nonce += hex_chars_small_h[rand() % 16];
        }
        return nonce;
}

void BisCuda::ResetAllDevices() {
        int nDev = GetNumDevices();
        for (int dev_id = 0; dev_id < nDev; dev_id++) {
                cur(hipSetDevice(dev_id));
                cur(hipDeviceReset());
        }
}

void BisCuda::Init() {
        n_devices = GetNumDevices();

        //Load MAP
        cur(hipHostMalloc(&map_host, map_size));
        printf("Loading heavy3a.bin into memory\n");
        FILE* in_file = fopen("heavy3a.bin", "rb");
        if (in_file != NULL) {
                fread(map_host, BIN_SIZE, 1, in_file);
                fclose(in_file);
                printf("Loaded heavy3a.bin");
        }
        else {
                printf("Could not load heavy3a.bin, exiting..");
                exit(0);
        }
        //EO load map

        //This is only first time "random" initial values, will be overwritten by the next call to Update
        Update("ddf74f55fb386d29037b010966d2424268a5efc6c8be2b8c6c9c9de9", "25293edc281c3de8ecd41047ea01288603c5aafa13f8f73c15b7871", "7cd4ab66b1a");

        for (uint32_t dev_id = 0; dev_id < n_devices; dev_id++) {
                InitDevice(dev_id);
        }
}

void BisCuda::InitDevice(uint32_t dev_id) {
        cur(hipSetDevice(dev_id));

        cur(hipMalloc(&mining_cond_uint_dev[dev_id], mining_cond_uint_size));
        cur(hipMalloc(&nonce_found_on_th_id_dev[dev_id], sizeof(uint32_t)));
        cur(hipMalloc(&hash_dev[dev_id], m_h_size));
        cur(hipMalloc(&message_int_dev[dev_id], msg_int_size));

        cur(hipHostMalloc((void**)&nonce_found_on_th_id_host[dev_id], sizeof(uint32_t)));
        cur(hipHostMalloc((void**)&message_int_host[dev_id], msg_int_size));
        cur(hipHostMalloc((void**)&hash_host[dev_id], msg_int_size));

        //Map
        cur(hipMalloc(&map_dev[dev_id], map_size));
        cur(hipMemcpy(map_dev[dev_id], map_host, map_size, hipMemcpyHostToDevice));

}

void BisCuda::DeletePointers() {
        for (uint32_t i = 0; i < n_devices; i++) {
                // Device
                cur(hipFree(mining_cond_uint_dev[i]));
                cur(hipFree(message_int_dev[i]));
                cur(hipFree(hash_dev[i]));
                cur(hipFree(nonce_found_on_th_id_dev[i]));

                // Host
                cur(hipHostFree(nonce_found_on_th_id_host[i]));
                cur(hipHostFree(message_int_host[i]));
                cur(hipHostFree(hash_host[i]));
        }
}

void BisCuda::Update(std::string address, std::string db_block_hash, std::string mining_condition_hex_string) {
        address_h = address;
        db_block_hash_h = db_block_hash;
        GenerateNonces();

        //Convert mining condition to int 32
        mining_cond_uint_host[0] = 0xFFFFFFFF;
        mining_cond_uint_host[1] = 0xFFFFFFFF;
        mining_condition_n_ints = StringToInts(mining_condition_hex_string, mining_cond_uint_host);
        tail_length_in_chars_host = 8 - (mining_condition_hex_string.length() % 8);
        if (tail_length_in_chars_host == 8) { tail_length_in_chars_host = 0; }

        for (uint32_t dev_id = 0; dev_id < n_devices; dev_id++) {
                shouldUpdateDevices[dev_id] = true;
        }
}

void BisCuda::UpdateDevice(uint32_t dev_id, bool set_device = true) {
        if (set_device) {
                cur(hipSetDevice(dev_id));
        }

        //Update message with new noce (this must be done each loop in order to find new solutions on a new message)
        nonce_h[dev_id] = GetNonce();
        message_string[dev_id] = address_h + nonce_h[dev_id] + db_block_hash_h;

        if (shouldUpdateDevices[dev_id]) {
                cur(hipMemcpy(mining_cond_uint_dev[dev_id], mining_cond_uint_host, mining_cond_uint_size, hipMemcpyHostToDevice));
                shouldUpdateDevices[dev_id] = false;
        }

        packMsg((const unsigned char*)message_string[dev_id].c_str(), message_int_host[dev_id]);
        sha224_init(hash_host[dev_id]);
        transform_cpu(message_int_host[dev_id], hash_host[dev_id]);
        cur(hipMemcpy(message_int_dev[dev_id], message_int_host[dev_id], msg_int_size, hipMemcpyHostToDevice));
        cur(hipMemcpy(hash_dev[dev_id], hash_host[dev_id], m_h_size, hipMemcpyHostToDevice));
}

void BisCuda::StartAllGPUs() {
        run_threads = true;
        for (uint32_t dev_id = 0; dev_id < n_devices; dev_id++) {
                threads[dev_id] = std::thread(&BisCuda::LoopDeviceThread, this, dev_id);
        }
}

void BisCuda::LoopDeviceThread(uint32_t dev_id) {
        cur(hipSetDevice(dev_id));
        while (run_threads) {
                //Update device each loop, in order to generate new nonces and messages
                UpdateDevice(dev_id, false);
                LoopDevice(dev_id, false);
                n_hashes_executed += N_THREADS;
        }
}

void BisCuda::LoopDevice(uint32_t dev_id, bool set_device = true) {
        if (set_device) {
                cur(hipSetDevice(dev_id));
        }

        *nonce_found_on_th_id_host[dev_id] = 0xFFFFFFFF;
        cur(hipMemcpy(nonce_found_on_th_id_dev[dev_id], nonce_found_on_th_id_host[dev_id], sizeof(uint32_t), hipMemcpyHostToDevice));

        sha224_find << <N_BLOCKS, N_THREADS_PER_BLOCK >> > (message_int_dev[dev_id], hash_dev[dev_id], mining_cond_uint_dev[dev_id], tail_length_in_chars_host, map_dev[dev_id], nonce_found_on_th_id_dev[dev_id]);

        hipDeviceSynchronize();

        cur(hipMemcpy(nonce_found_on_th_id_host[dev_id], nonce_found_on_th_id_dev[dev_id], sizeof(uint32_t), hipMemcpyDeviceToHost));

        if (*nonce_found_on_th_id_host[dev_id] != 0xFFFFFFFF) {
                std::string validNonce = nonceFromThreadId(nonce_h[dev_id], *nonce_found_on_th_id_host[dev_id]) + "\n";
                ValidNoncesAdd(validNonce);
                std::cout << "\nNonce found!" << std::flush;
        }
}
